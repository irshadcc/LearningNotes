#include "hip/hip_runtime.h"
#include "blur.hpp"
#include "cuda_util.h"
#include <hip/driver_types.h>
#include <stdexcept>
#include <string>

__constant__ int d_rows;
__constant__ int d_columns;


__device__ int get1DIdxRowMajor(int x, int y, int rows, int columns) {
  return y*columns + x;
} 


template<int blockSize>
__global__ void BlurKernel(int* r, int *g, int *b) {

    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x >= d_columns || y > d_rows) {
        return ;
    }

    __shared__ int data_r[blockSize*blockSize]; 
    __shared__ int data_g[blockSize*blockSize]; 
    __shared__ int data_b[blockSize*blockSize]; 

    int block1DIdx = 0; // Complete
    int idx = get1DIdxRowMajor(x, y, d_rows, d_columns);
    data_r[block1DIdx] = r[idx]; 

    // Sync all threads in block
    __syncthreads();




}


void BlurKernelCuda(int *r, int *g, int *b, int rows, int columns) {
  constexpr int BlockSize =3;

  int npixels = rows * columns;
  int *d_r_image, *d_g_image, *d_b_image;

  CUDA_CHECK(hipMalloc(&d_r_image, npixels), 
    "Failed to allocate for memory for Red pixels");
  CUDA_CHECK(hipMalloc(&d_g_image, npixels), 
    "Failed to allocate for memory for Green pixels");
  CUDA_CHECK(hipMalloc(&d_b_image, npixels), 
    "Failed to allocate for memory for Blue pixels");

  CUDA_CHECK(hipMemcpy(d_r_image, r, npixels, hipMemcpyHostToDevice),
    "Failed to copy red pixels to device");
  CUDA_CHECK(hipMemcpy(d_g_image, g, npixels, hipMemcpyHostToDevice),
    "Failed to copy green pixels to device");
  CUDA_CHECK(hipMemcpy(d_b_image, b, npixels, hipMemcpyHostToDevice),
    "Failed to copy blue pixels to device");

  // Copy symbols

  CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_rows), &rows, 1), "Failed to copy rows");
  CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_columns), &columns, 1),
             "Failed to copy columns");

  dim3 gridDim(std::ceil(columns / BlockSize), std::ceil(rows / BlockSize));
  dim3 blockDim(3, 3);

  int Gx = std::ceil(columns / 3.0);
  int Gy = std::ceil(rows / 3.0);

  BlurKernel<3><<<gridDim, blockDim>>>(d_r_image, d_g_image, d_b_image);

  CUDA_CHECK(hipMemcpy(r, d_r_image, npixels, hipMemcpyDeviceToHost),
    "Failed to copy red pixels from device");
  CUDA_CHECK(hipMemcpy(g, d_g_image, npixels, hipMemcpyDeviceToHost),
    "Failed to copy green pixels from device");
  CUDA_CHECK(hipMemcpy(b, d_b_image, npixels, hipMemcpyDeviceToHost),
    "Failed to copy blue pixels from device");

  CUDA_CHECK(hipFree(d_r_image), "Failed to deallocate red pixels from device memory");
  CUDA_CHECK(hipFree(d_g_image), "Failed to deallocate green pixels from device memory");
  CUDA_CHECK(hipFree(d_b_image), "Failed to deallocate blue pixels from device memory");

}