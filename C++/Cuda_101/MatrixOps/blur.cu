#include "hip/hip_runtime.h"
#include "blur.hpp"
#include "cuda_util.h"
#include <stdexcept>
#include <string>


__constant__ int d_rows ; 
__constant__ int d_columns;

__global__ void BlurKernel(int* image_in, int* image_out, int rows, int columns) {

    image_out[0] = 0;

}

int* BlurKernelCuda(int *image, int rows, int columns) {

    int npixels = rows*columns;
    int *d_image_in, *d_image_out;

    int *blurred_image = new int[npixels];

    CUDA_CHECK(hipMalloc(&d_image_in, npixels), 
        "Failed to allocate memory for image");
    CUDA_CHECK(hipMalloc(&d_image_out, npixels), 
        "Failed to allocate memory for image");

    CUDA_CHECK(hipMemcpy(d_image_in, image, npixels, hipMemcpyHostToDevice), 
        "Failed to copy image to device");

    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_rows), &rows, 1), 
        "Failed to copy rows");
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_columns), &columns, 1), 
        "Failed to copy columns");
    
    dim3 gridDim(
        std::ceil(columns/3.0),
        std::ceil(rows/3.0)
    );
    dim3 blockDim(1, 3, 3);

    int Gx = std::ceil(columns/3.0);
    int Gy = std::ceil(rows/3.0);

    BlurKernel<<<gridDim, blockDim>>>(d_image_in, d_image_out, rows, columns);
    CUDA_CHECK(hipMemcpy(blurred_image, d_image_out, npixels, hipMemcpyDeviceToHost), 
        "Failed to copy image from device to host");
    return blurred_image;
}