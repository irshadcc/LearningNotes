
#include <hip/hip_runtime.h>

__global__ void matrixMultiply(int** matrixA, int m, int n, int** matrixB,
                               int p, int q, int** matrixC) {
  int xIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int yIdx = blockIdx.y * blockDim.y + threadIdx.y;
  matrixC[xIdx][yIdx] = matrixA[xIdx][yIdx] + matrixB[xIdx][yIdx];
}